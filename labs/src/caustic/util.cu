#include "hip/hip_runtime.h"
#include <stdint.h>
#include <stdlib.h>
#include <string.h>

#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>
#include <cutil.h>

#include "util.h"

Matrix init_matrix(const unsigned hgt, const unsigned wid) {
    unsigned n_bytes = hgt * wid * sizeof(float);

    Matrix out;
    out.hgt = hgt;
    out.wid = wid;

    hipMalloc((void **)&out.elems, n_bytes);
    hipMemset(out.elems, 0, n_bytes);

    return out;
}

Matrix to_device(const Matrix m) {
    Matrix m_d = m;
    size_t len = m.wid * m.hgt * sizeof(float);
    hipMalloc((void **)&m_d.elems, len);
    hipMemcpy(m_d.elems, m.elems, len, hipMemcpyHostToDevice);
    return m_d;
}

__global__ void dev_init_mesh(float *x, float *y, float *z) {
    unsigned i = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned j = blockIdx.y * blockDim.y + threadIdx.y;

    if (!(i < IMG_DIM && j < IMG_DIM)) return;

    unsigned idx = FLAT(i, j, IMG_DIM);
    x[idx] = j;
    y[idx] = IMG_DIM - i - 1;
    z[idx] = 0;
}

Mesh init_mesh(const unsigned hgt, const unsigned wid) {
    unsigned n_bytes = hgt * wid * sizeof(float);

    Mesh out;
    out.hgt = hgt;
    out.wid = wid;

    hipMalloc((void **)&out.x, n_bytes);
    hipMalloc((void **)&out.y, n_bytes);
    hipMalloc((void **)&out.z, n_bytes);

    dim3 dimGrid(N_BLK(out.hgt, BLKSIZE_2D), N_BLK(out.wid, BLKSIZE_2D));
    dim3 dimBlk(BLKSIZE_2D, BLKSIZE_2D);
    dev_init_mesh<<<dimGrid, dimBlk>>>(out.x, out.y, out.z);
    hipDeviceSynchronize();

    return out;
}