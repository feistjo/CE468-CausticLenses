#include <stdint.h>
#include <stdlib.h>
#include <stdio.h>
#include <iostream>

#include <hipblas.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <cutil.h>

#include "util.h"
#include "kernels.h"

void optimize_mesh(Matrix img, Mesh mesh, Matrix loss, hipblasHandle_t ch) {
    unsigned len = img.hgt * img.wid;

    compute_loss(mesh, img, loss);

    // normalize loss: loss = loss - (sum(loss) / len)
    float loss_sum = 0; 
    hipblasSasum(ch, len, loss.elems, 1, &loss_sum);
    scalar_add(loss, -loss_sum / len, loss);

    Matrix phi = init_matrix(img.hgt, img.wid);

    // run until convergence
    // TODO: only using 1 iteration right now
    for (unsigned i = 0; i < 1; i++) {
        float max_update = relax(phi, loss, ch);

        printf("[%d]: Relaxed phi by %f\n", i, max_update);

        if (max_update < 0.00001) {
            printf("[%d]: Converged to optimal phi\n", i);
            break;
        }
    }

    // march mesh based on converged phi
}

int create_mesh(Matrix host_img) {
    // CUBLAS stuff
    hipblasHandle_t ch = NULL;
    hipStream_t stream = NULL;
    hipblasCreate(&ch);
    hipblasSetStream(ch, stream);

    unsigned len = host_img.hgt * host_img.wid;

    Matrix img = to_device(host_img);

    // The mesh stores the 3D position of each point on the lens
    Mesh mesh = init_mesh(img.hgt + 1, img.wid + 1);
    
    // calculate sums
    float mesh_sum = float(len);
    float img_sum = 0;
    hipblasSasum(ch, len, img.elems, 1, &img_sum);

    // boost brightness of image
    float boost_ratio = mesh_sum / img_sum;
    hipblasSscal(ch, len, &boost_ratio, img.elems, 1);

    Matrix loss = init_matrix(host_img.hgt, host_img.wid);

    // optimize mesh until convergence
    // TODO: using only 1 iteration right now
    optimize_mesh(img, mesh, loss, ch);

    /*
    float artifact_size = 0.1;
    float focal_length = 0.2;

    Matrix heights_d;
    // call kernel to find surface of mesh (heights)
    // call kernel to create bottom of mesh (solidify)
    // write output file obj
    */

    hipFree(img.elems);
    hipFree(loss.elems);
    return 0;
}